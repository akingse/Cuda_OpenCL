#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <vector>
//non-support array
#include <Eigen/Dense>
#include "calculateOccultedCuda.h"
//#include "clashTypeDefine.h" //using container vector
using namespace Eigen;
using namespace cuda;

__global__ void isTwoTrianglesIntersectSAT_P(bool* isinter, const Triangle2d* triA, const Triangle2d* triB)
{
	Eigen::Vector2d edgesAB[6] = {
		triA->data[1] - triA->data[0],
		triA->data[2] - triA->data[1],
		triA->data[0] - triA->data[2],
		triB->data[1] - triB->data[0],
		triB->data[2] - triB->data[1],
		triB->data[0] - triB->data[2],
	};
	for (int i = 0; i < 6; i++)
		edgesAB[i] = { -edgesAB[i][1], edgesAB[i][0] }; //rotz(pi/2)
	double minA, maxA, minB, maxB, projection;
	for (int i = 0; i < 6; i++)
	{
		//if (axis.isZero()) //degeneracy triangle, regard as not shield
		//	continue;
		minA = DBL_MAX;
		maxA = -DBL_MAX;
		minB = DBL_MAX;
		maxB = -DBL_MAX;
		for (int j = 0; j < 3; j++)
		{
			projection = edgesAB[i].dot(triA->data[j]);
			minA = min(minA, projection);
			maxA = max(maxA, projection);
			projection = edgesAB[i].dot(triB->data[j]);
			minB = min(minB, projection);
			maxB = max(maxB, projection);
		}
		if (maxA <= minB || maxB <= minA) //contact, regard as not shield
		{
			*isinter = false;
			printf("sat=false.");
			return;
		}
	}
	*isinter = true;
	printf("sat=true.");
}

//#define USING_POINTER
static void test_triangle()
{
	const int N = 1;
#ifdef USING_POINTER
	Triangle2d* triA, *triB;
	hipMallocManaged(&triA, N * 6 * sizeof(Triangle2d));
	hipMallocManaged(&triB, N * 6 * sizeof(Triangle2d));
	for (int i = 0; i < N; i++) 
	{
		triA[i] = Triangle2d{
				Vector2d{0,0},
				Vector2d{2,0},
				Vector2d{0,1},
		};
		triB[i] = Triangle2d{
				Vector2d{1,0},
				Vector2d{2,0},
				Vector2d{2,1},
		};
	}

	bool* isinter; //gpu mem
	hipMallocManaged(&isinter, N * sizeof(bool));
	//kernal
	isTwoTrianglesIntersectSAT_P << <1, N >> > (isinter, triA, triB);
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	//bool* res;
	bool res[N]; //cpu mem
	hipError_t cudaStatus = hipMemcpy(res, isinter, N * sizeof(bool), hipMemcpyDeviceToHost);
#else
	std::vector<Triangle2d> triA(N), triB(N);
	for (int i = 0; i < N; i++) 
	{
		triA[i] = Triangle2d{
				Vector2d{0,0},
				Vector2d{2,0},
				Vector2d{0,1},
		};
		triB[i] = Triangle2d{
				Vector2d{1,0},
				Vector2d{2,0},
				Vector2d{2,1},
		};
	}
	Triangle2d* dev_a; //device var
	Triangle2d* dev_b;
	bool* dev_c;
	hipMallocManaged(&dev_a, N * sizeof(Triangle2d));
	hipMallocManaged(&dev_b, N * sizeof(Triangle2d));
	hipMallocManaged(&dev_c, N * sizeof(bool));
	hipMemcpy(dev_a, triA.data(), N * sizeof(Triangle2d), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, triB.data(), N * sizeof(Triangle2d), hipMemcpyHostToDevice);
	isTwoTrianglesIntersectSAT_P << <1, N >> > (dev_c, dev_a, dev_b);
	hipDeviceSynchronize();
	//copy out from gpu
	bool h_isinter[N];
	hipMemcpy(h_isinter, dev_c, N * sizeof(bool), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
#endif
	return;
}

static int _enrol = []()
    {
        //test_triangle();
		//test_vector();
        return 0;
    }();
