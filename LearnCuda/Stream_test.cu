#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
using namespace std;

const int N = 1 << 20;

__global__ void kernel_pi(float* x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159, i));
    }
}

int main_5()
{
    const int num_streams = 8;

    hipStream_t streams[num_streams];
    float* data[num_streams];

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);

        hipMalloc(&data[i], N * sizeof(float));

        // launch one worker kernel per stream
        kernel_pi << <1, 64, 0, streams[i] >> > (data[i], N);

        // launch a dummy kernel on the default stream
        kernel_pi << <1, 1 >> > (0, 0);
    }

    hipDeviceReset();

    return 0;
}